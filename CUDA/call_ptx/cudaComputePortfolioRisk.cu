/*
nyck33@lenovo-gtx1650:/mnt/d/LLVM/Lean/CUDA$ nvcc -o testNoCublasCovarMatrix noCublasDirver.cpp cudaComputePortfolioRisk.cu -lcudart
nyck33@lenovo-gtx1650:/mnt/d/LLVM/Lean/CUDA$ ./testNoCublasCovarMatrix      
GPU computation time: 1.01582 seconds.
CPU computation time: 8.46907 seconds.
Results are correct.
*/

#include <hip/hip_runtime.h>



// CUDA kernel to perform the matrix multiplication A * B
__global__ void matrixMultiply(double* A, double* B, double* C, int ARows, int ACols, int BCols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < ARows && col < BCols) {
        double sum = 0.0;
        for (int k = 0; k < ACols; ++k) {
            sum += A[row * ACols + k] * B[k * BCols + col];
        }
        C[row * BCols + col] = sum;
    }
}

// Host function to initialize memory, call the kernels, and clean up
extern "C" void computeCovarianceMatrix(double* S, double* R, double* Sigma, int sRows, int sCols) {
    double *d_S, *d_R, *d_T, *d_Sigma;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&d_S, sRows * sCols * sizeof(double));
    hipMallocManaged(&d_R, sCols * sCols * sizeof(double));
    hipMallocManaged(&d_T, sRows * sCols * sizeof(double)); // Intermediate result
    hipMallocManaged(&d_Sigma, sRows * sRows * sizeof(double));

    // Copy data into managed memory
    hipMemcpy(d_S, S, sRows * sCols * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_R, R, sCols * sCols * sizeof(double), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid1((sCols + 15) / 16, (sRows + 15) / 16);
    dim3 blocksPerGrid2((sRows + 15) / 16, (sRows + 15) / 16);

    // Perform S * R = T
    matrixMultiply<<<blocksPerGrid1, threadsPerBlock>>>(d_S, d_R, d_T, sRows, sCols, sCols);

    // Perform T * S^T = Sigma (assuming S is square and sCols == sRows)
    matrixMultiply<<<blocksPerGrid2, threadsPerBlock>>>(d_T, d_S, d_Sigma, sRows, sCols, sRows);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Copy the result matrix back to the host memory
    hipMemcpy(Sigma, d_Sigma, sRows * sRows * sizeof(double), hipMemcpyDeviceToHost);

    // Free the device memory
    hipFree(d_S);
    hipFree(d_R);
    hipFree(d_T);
    hipFree(d_Sigma);
}
