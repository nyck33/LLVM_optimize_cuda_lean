#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    hipModule_t cuModule;
    hipFunction_t cuFunction;
    hipInit(0);
    hipDevice_t cuDevice;
    hipDeviceGet(&cuDevice, 0);
    hipCtx_t cuContext;
    hipCtxCreate(&cuContext, 0, cuDevice);
    hipModuleLoad(&cuModule, "cudaComputePortfolioRisk.ptx");
    hipModuleGetFunction(&cuFunction, cuModule, "_Z14matrixMultiplyPdS_S_iii");

    // Set up kernel parameters and launch the kernel
    // ...

    hipCtxDestroy(cuContext);
    return 0;
}