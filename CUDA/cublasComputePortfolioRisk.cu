//nvcc -arch=sm_75 --compiler-options '-fPIC' -shared -o libcovariance.so cublasComputePortfolioRisk.cu -lcublas

#include <hip/hip_runtime.h>
#include <hipblas.h>

extern "C" {
    void computeCovarianceMatrix(double* s, double* r, double* result, int sRows, int sCols, int rCols, hipStream_t stream = 0) {
        double* d_s; // Pointer for the device memory of matrix S
        double* d_r; // Pointer for the device memory of matrix R
        double* d_result; // Pointer for the device memory of the result matrix
        double* d_temp; // Temporary device memory to hold intermediate results

        // Allocate device memory for matrices S, R, and the result
        hipMalloc(&d_s, sRows * sCols * sizeof(double));
        hipMalloc(&d_r, sCols * rCols * sizeof(double));
        hipMalloc(&d_result, sRows * rCols * sizeof(double));
        hipMalloc(&d_temp, sRows * rCols * sizeof(double)); // Memory for the intermediate result S * R

        // Copy matrices S and R from host to device memory
        hipMemcpy(d_s, s, sRows * sCols * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_r, r, sCols * rCols * sizeof(double), hipMemcpyHostToDevice);

        // Create a handle for cuBLAS operations
        hipblasHandle_t handle;
        hipblasCreate(&handle);
        if (stream) {
            hipblasSetStream(handle, stream);
        }

        // Define scalar values for the multiplication
        const double alpha = 1.0;
        const double beta = 0.0;


        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rCols, sRows, sCols, &alpha, d_r, rCols, d_s, sCols, &beta, d_temp, rCols);
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, sRows, sRows, rCols, &alpha, d_temp, sRows, d_s, sCols, &beta, d_result, sRows);
        
        /*
        // Perform matrix multiplication result = d_temp * S^T
        cublasDgemm(handle, CUBLAS_OP_N, CUBLAS_OP_T,
                    sRows, rCols, sCols, &alpha,
                    d_s, sRows, d_r, sCols, &beta,
                    d_temp, sRows);

        // Perform matrix multiplication result = d_temp * S^T
        cublasDgemm(handle, CUBLAS_OP_N, CUBLAS_OP_T,
                    sRows, sRows, rCols, &alpha,
                    d_temp, sRows, d_s, sRows, &beta,
                    d_result, sRows);
        */
        // Copy the result matrix from device to host memory
        hipMemcpy(result, d_result, sRows * sRows * sizeof(double), hipMemcpyDeviceToHost);

        // Free the device memory allocated for matrices S, R, the result, and temporary storage
        hipFree(d_s);
        hipFree(d_r);
        hipFree(d_result);
        hipFree(d_temp);

        // Destroy the cuBLAS handle
        hipblasDestroy(handle);
    }
}
